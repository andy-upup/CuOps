#include <hip/hip_runtime.h>

#include <stdlib.h>

#include <cstdio>

#define THREAD_PER_BLOCK 256

__global__ void reduce_warp_divergence(const float* src, float* dst,
                                       const int N) {
  const int bx = blockIdx.x;
  const int tx = threadIdx.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float shared[THREAD_PER_BLOCK];
  shared[tx] = src[tid];
  __syncthreads();
  for (int i = 1; i < THREAD_PER_BLOCK; i *= 2) {
    if (tx < THREAD_PER_BLOCK / (2 * i)) {
      const int index = 2 * tx * i;
      shared[index] += shared[index + i];
    }
    __syncthreads();
  }
  if (tx == 0) {
    dst[bx] = shared[0];
  }
}

bool check(const float* output, const float* golden, const int N) {
  for (int i = 0; i < N; ++i) {
    if (std::abs(output[i] - golden[i]) >= 1e-4) {
      return false;
    }
  }
  return true;
}

int main() {
  const int N = 32 * 1024 * 1024;
  float* input = (float*)malloc(N * sizeof(float));
  float* d_input;
  hipMalloc((void**)&d_input, N * sizeof(float));

  const int num_block = N / THREAD_PER_BLOCK;
  float* output = (float*)malloc(num_block * sizeof(float));
  float* d_output;
  hipMalloc((void**)&d_output, num_block * sizeof(float));
  float* golden = (float*)malloc(num_block * sizeof(float));

  for (int i = 0; i < N; ++i) {
    input[i] = 2.0 * (float)drand48() - 1.0;
  }

  for (int i = 0; i < num_block; ++i) {
    float sum_block = 0.f;
    for (int j = 0; j < THREAD_PER_BLOCK; ++j) {
      sum_block += input[i * THREAD_PER_BLOCK + j];
    }
    golden[i] = sum_block;
  }

  hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 grid(num_block);
  dim3 block(THREAD_PER_BLOCK);
  reduce_warp_divergence<<<grid, block>>>(d_input, d_output, N);

  hipMemcpy(output, d_output, num_block * sizeof(float),
             hipMemcpyDeviceToHost);
  if (check(output, golden, num_block)) {
    printf("Output is right.\n");
  } else {
    printf("Output is wrong!\n");
    for (int i = 0; i < num_block; ++i) {
      printf("%lf", output[i]);
    }
    printf("\n");
  }

  hipFree(d_input);
  hipFree(d_output);
  free(output);
  free(golden);
  return 0;
}