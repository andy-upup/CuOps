#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdlib.h>

#include <cute/tensor.hpp>

#include "utils.hpp"

using namespace cute;

TEST(Cute, MakeLayout) {
  constexpr int M = 3;
  constexpr int N = 4;
  auto layout3x4 = make_layout(make_shape(Int<M>{}, Int<N>{}));
  PRINT("layout3x4", layout3x4);

  auto shape8 = make_shape(Int<8>{});
  auto stride1 = make_stride(Int<1>{});
  auto layout8_1 = make_layout(shape8, stride1);
  PRINT("layout8_1", layout8_1);

  auto shape4x5 = make_shape(Int<4>{}, Int<5>{});
  auto stride5x1 = make_stride(Int<5>{}, Int<1>{});
  auto layout4x5_5x1 = make_layout(shape4x5, stride5x1);
  PRINT("layout4x5_5x1", layout4x5_5x1);

  auto shape2x3x4 = make_shape(Int<2>{}, Int<3>{}, Int<4>{});
  auto stride12x4x1 = make_stride(Int<12>{}, Int<4>{}, Int<1>{});
  auto layout2x3x4_12x4x1 = make_layout(shape2x3x4, stride12x4x1);
  PRINT("layout2x3x4_12x4x1", layout2x3x4_12x4x1);
}

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}